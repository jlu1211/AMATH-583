#include <iostream>
#include <vector>
#include <complex>
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define CUDA_CHECK(call)                                                   \
  do {                                                                     \
    hipError_t err = call;                                                \
    if(err != hipSuccess) {                                               \
      std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__         \
                << " code=" << err << " \""                                 \
                << hipGetErrorString(err) << "\"\n";                      \
      std::exit(EXIT_FAILURE);                                             \
    }                                                                      \
  } while(0)

#define CUFFT_CHECK(call)                                                  \
  do {                                                                     \
    hipfftResult cres = call;                                               \
    if(cres != HIPFFT_SUCCESS) {                                            \
      std::cerr << "cuFFT error at " << __FILE__ << ":" << __LINE__        \
                << " code=" << cres << "\n";                               \
      std::exit(EXIT_FAILURE);                                             \
    }                                                                      \
  } while(0)

void initializePlaneWave(std::complex<double>* phi, int n) {
    const double TWO_PI = 2.0 * M_PI;
    int kx = 1, ky = 2, kz = 3;

    for(int iz = 0; iz < n; iz++) {
        for(int iy = 0; iy < n; iy++) {
            for(int ix = 0; ix < n; ix++) {
                size_t idx = (size_t)iz * n * n + (size_t)iy * n + (size_t)ix;
                double x = double(ix) * (TWO_PI / double(n));
                double y = double(iy) * (TWO_PI / double(n));
                double z = double(iz) * (TWO_PI / double(n));
                double phase = kx*x + ky*y + kz*z;
                phi[idx] = std::complex<double>(cos(phase), sin(phase));
            }
        }
    }
}

inline double flopEstimate(int n) {
    return 360.0 * double(n)*double(n)*double(n) * std::log2(double(n));
}

__global__
void multiplyByIKx(hipfftDoubleComplex* devPhi,
                   hipfftDoubleComplex* devGx,
                   int n, double scale)
{
    size_t idx  = blockIdx.x * blockDim.x + threadIdx.x;
    size_t N    = size_t(n) * n * n;
    if(idx >= N) return;

    size_t tmp = idx;
    int ix = tmp % n;  tmp /= n;
    int iy = tmp % n;  tmp /= n;
    int iz = tmp;

    int kx = (ix <= n/2) ? ix : ix - n;

    hipfftDoubleComplex ph = devPhi[idx];
    devGx[idx].x = scale * (-double(kx) * ph.y);
    devGx[idx].y = scale * ( double(kx) * ph.x);
}

__global__
void multiplyByIKy(hipfftDoubleComplex* devPhi,
                   hipfftDoubleComplex* devGy,
                   int n, double scale)
{
    size_t idx  = blockIdx.x * blockDim.x + threadIdx.x;
    size_t N    = size_t(n) * n * n;
    if(idx >= N) return;

    size_t tmp = idx;
    int ix = tmp % n;  tmp /= n;
    int iy = tmp % n;  tmp /= n;
    int iz = tmp;

    int ky = (iy <= n/2) ? iy : iy - n;
    hipfftDoubleComplex ph = devPhi[idx];
    devGy[idx].x = scale * (-double(ky) * ph.y);
    devGy[idx].y = scale * ( double(ky) * ph.x);
}

__global__
void multiplyByIKz(hipfftDoubleComplex* devPhi,
                   hipfftDoubleComplex* devGz,
                   int n, double scale)
{
    size_t idx  = blockIdx.x * blockDim.x + threadIdx.x;
    size_t N    = size_t(n) * n * n;
    if(idx >= N) return;

    size_t tmp = idx;
    int ix = tmp % n;  tmp /= n;
    int iy = tmp % n;  tmp /= n;
    int iz = tmp;

    int kz = (iz <= n/2) ? iz : iz - n;
    hipfftDoubleComplex ph = devPhi[idx];
    devGz[idx].x = scale * (-double(kz) * ph.y);
    devGz[idx].y = scale * ( double(kz) * ph.x);
}

double runCuFFTGradient(int n, int ntrial) {
    size_t N = size_t(n) * n * n;
    size_t bytes = sizeof(hipfftDoubleComplex) * N;

    std::complex<double>* HostCopy =
        reinterpret_cast<std::complex<double>*>(malloc(sizeof(std::complex<double>) * N));
    initializePlaneWave(HostCopy, n);

    hipfftDoubleComplex *devPhi, *devGx, *devGy, *devGz;
    CUDA_CHECK(hipMalloc(&devPhi, bytes));
    CUDA_CHECK(hipMalloc(&devGx,  bytes));
    CUDA_CHECK(hipMalloc(&devGy,  bytes));
    CUDA_CHECK(hipMalloc(&devGz,  bytes));

    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan3d(&plan, n, n, n, HIPFFT_Z2Z));

    hipEvent_t startEvent, stopEvent;
    CUDA_CHECK(hipEventCreate(&startEvent));
    CUDA_CHECK(hipEventCreate(&stopEvent));

    int blockSize = 256;
    int gridSize  = int((N + blockSize - 1) / blockSize);
    double scale  = 1.0 / double(N);

    double total_time_ms = 0.0;
    for(int trial = 0; trial < ntrial; trial++) {
        CUDA_CHECK(hipMemcpy(devPhi, HostCopy, bytes, hipMemcpyHostToDevice));

        CUDA_CHECK(hipEventRecord(startEvent, 0));

        CUFFT_CHECK(hipfftExecZ2Z(plan, devPhi, devPhi, HIPFFT_FORWARD));

        multiplyByIKx<<<gridSize, blockSize>>>(devPhi, devGx, n, scale);
        multiplyByIKy<<<gridSize, blockSize>>>(devPhi, devGy, n, scale);
        multiplyByIKz<<<gridSize, blockSize>>>(devPhi, devGz, n, scale);

        CUFFT_CHECK(hipfftExecZ2Z(plan, devGx, devGx, HIPFFT_BACKWARD));
        CUFFT_CHECK(hipfftExecZ2Z(plan, devGy, devGy, HIPFFT_BACKWARD));
        CUFFT_CHECK(hipfftExecZ2Z(plan, devGz, devGz, HIPFFT_BACKWARD));

        CUDA_CHECK(hipEventRecord(stopEvent, 0));
        CUDA_CHECK(hipEventSynchronize(stopEvent));
        float elapsed_ms = 0.0f;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, startEvent, stopEvent));
        total_time_ms += double(elapsed_ms);
    }

    double avg_time_s = (total_time_ms / double(ntrial)) / 1000.0;

    hipfftDestroy(plan);
    hipFree(devPhi);
    hipFree(devGx);
    hipFree(devGy);
    hipFree(devGz);
    free(HostCopy);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return avg_time_s;
}

int main() {
    std::cout << "n,avg_time_cufft_s,gflops_cufft\n";

    std::vector<int> ns = {16, 32, 64, 128, 256};
    int ntrial = 3;

    for(int n : ns) {
        double t_cufft = runCuFFTGradient(n, ntrial);
        double flops   = flopEstimate(n);
        double ggpu    = flops / (t_cufft * 1e9);

        std::printf("%d,%.6f,%.2f\n", n, t_cufft, ggpu);
    }

    return 0;
}